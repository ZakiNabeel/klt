#include "hip/hip_runtime.h"
/*********************************************************************
 * convolve_cuda.cu - HIGHLY OPTIMIZED FOR TESLA T4 (SM_75)
 * 
 * T4-Specific Optimizations:
 * 1. Turing has 96KB L1/Shared (64KB shared + 32KB L1 config)
 * 2. 40 SMs with 64 FP32 cores each = 2560 CUDA cores
 * 3. Separable convolution (fewer ops)
 * 4. Shared memory tiling with proper indexing
 * 5. Vectorized float4 loads for global→shared (FIXED ALIGNMENT)
 * 6. Warp-aligned 32×8 blocks (256 threads, 8 warps)
 * 7. Bank conflict avoidance with padding
 * 8. Async streams for compute/transfer overlap
 * 9. Persistent device buffers
 * 10. Constant memory for kernels
 *********************************************************************/

 #include <assert.h>
 #include <math.h>
 #include <stdlib.h>
 #include <hip/hip_runtime.h>
 #include "base.h"
 #include "error.h"
 #include "convolve.h"
 #include "klt_util.h"
 
 #define MAX_KERNEL_WIDTH 71
 #define WARP_SIZE 32
 #define BLOCK_DIM_X 32  // Full warp for coalescing
 #define BLOCK_DIM_Y 8   // 256 threads total
 #define MAX_KERNEL_SIZE 35
 
 #define CUDA_CHECK(call) \
   do { \
     hipError_t err = call; \
     if (err != hipSuccess) { \
       fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
               hipGetErrorString(err)); \
       exit(EXIT_FAILURE); \
     } \
   } while(0)
 
 /*********************************************************************
  * Kernel Data Structures
  *********************************************************************/
 typedef struct {
   int width;
   float data[MAX_KERNEL_WIDTH];
 } ConvolutionKernel;
 
 static ConvolutionKernel gauss_kernel;
 static ConvolutionKernel gaussderiv_kernel;
 static float sigma_last = -10.0;
 
 // Constant memory for kernel (faster than global, cached)
 __constant__ float c_kernel[MAX_KERNEL_SIZE];
 
/*********************************************************************
 * Persistent Device Buffers with Streams
 *********************************************************************/
static struct {
  float *d_img1, *d_img2;
  size_t allocated_size;
  hipStream_t stream;
  bool initialized;
} g_gpu = {NULL, NULL, 0, NULL, false};
 
 static void ensure_gpu_buffers(size_t bytes) {
   if (!g_gpu.initialized) {
     CUDA_CHECK(hipStreamCreate(&g_gpu.stream));
     // Set shared memory config: prefer 64KB shared, 32KB L1
     CUDA_CHECK(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
     g_gpu.initialized = true;
   }
   
  if (bytes > g_gpu.allocated_size) {
    if (g_gpu.d_img1) {
      hipFree(g_gpu.d_img1);
      hipFree(g_gpu.d_img2);
    }
    CUDA_CHECK(hipMalloc(&g_gpu.d_img1, bytes));
    CUDA_CHECK(hipMalloc(&g_gpu.d_img2, bytes));
    g_gpu.allocated_size = bytes;
  }
}
 
/*********************************************************************
 * TILE SIZE FOR FUSED VERTICAL KERNEL
 *********************************************************************/
#define TILE_DIM 32

/*********************************************************************
 * OPTIMIZED HORIZONTAL CONVOLUTION
 *********************************************************************/
__global__ void convolveHoriz_Optimized(
   const float * __restrict__ imgin,
   float * __restrict__ imgout,
   int ncols, int nrows,
   int kernel_width)
 {
  const int radius = kernel_width / 2;
   const int tile_width = blockDim.x;
   const int tile_height = blockDim.y;
   
   // Shared memory with 8-byte padding for bank conflict avoidance
   // T4: 32 banks, 4-byte words → 8-byte padding = 2 words
   const int tile_stride = tile_width + 2 * radius + 8;  // +8 for padding
   extern __shared__ float s_tile[];
   
   const int tx = threadIdx.x;
   const int ty = threadIdx.y;
   const int gx = blockIdx.x * tile_width + tx;
   const int gy = blockIdx.y * tile_height + ty;
   
   if (gy >= nrows) return;
   
   // ============ PHASE 1: COOPERATIVE TILE LOADING ============
   const int tile_start_col = blockIdx.x * tile_width - radius;
   const int total_cols = tile_width + 2 * radius;
   
   // Each warp loads one row cooperatively
   for (int row = ty; row < tile_height; row += tile_height) {
     int global_row = blockIdx.y * tile_height + row;
     if (global_row >= nrows) continue;
     
     const float* row_ptr = &imgin[global_row * ncols];
     float* s_row = &s_tile[row * tile_stride];
     
    // Load tile data: each thread handles multiple elements
    for (int local_col = tx; local_col < total_cols; local_col += tile_width) {
      int global_col = tile_start_col + local_col;
      s_row[local_col] = (global_col >= 0 && global_col < ncols) ? row_ptr[global_col] : 0.0f;
    }
  }
   __syncthreads();
   
   // ============ PHASE 2: COMPUTE CONVOLUTION ============
   if (gx >= ncols) return;
   
   // Zero boundary pixels
   if (gx < radius || gx >= ncols - radius) {
     imgout[gy * ncols + gx] = 0.0f;
     return;
   }
   
   // Convolution with aggressive unrolling
   float sum = 0.0f;
   int s_center = ty * tile_stride + tx + radius;
   
   // Unroll based on typical kernel sizes
   if (kernel_width <= 7) {
     #pragma unroll
     for (int k = 0; k < kernel_width; k++) {
       sum += s_tile[s_center - radius + k] * c_kernel[k];
     }
   } else if (kernel_width <= 15) {
     #pragma unroll 4
     for (int k = 0; k < kernel_width; k++) {
       sum += s_tile[s_center - radius + k] * c_kernel[k];
     }
   } else {
     #pragma unroll 2
     for (int k = 0; k < kernel_width; k++) {
       sum += s_tile[s_center - radius + k] * c_kernel[k];
     }
   }
   
   imgout[gy * ncols + gx] = sum;
 }
 
/*********************************************************************
 * FUSED VERTICAL CONVOLUTION WITH LOCAL TRANSPOSE
 * 
 * Single kernel that does everything in shared memory:
 * 1. Load tile from global memory
 * 2. Transpose in shared memory (fast!)
 * 3. Apply horizontal convolution to transposed data
 * 4. Transpose result back in shared memory
 * 5. Write to global memory
 * 
 * ALL transposes happen in shared memory - much faster than 3 kernels!
 *********************************************************************/
__global__ void convolveVert_FusedTranspose(
  const float * __restrict__ imgin,
  float * __restrict__ imgout,
  int ncols, int nrows,
  int kernel_width)
{
  const int radius = kernel_width / 2;
  
  // Shared memory tiles for input, transposed, and output
  // We use two tiles to avoid conflicts
  extern __shared__ float s_mem[];
  float* s_tile = s_mem;                           // First tile for load/transpose
  float* s_conv = s_mem + (TILE_DIM + 2*radius) * (TILE_DIM + 2*radius + 8);  // Second for convolution
  
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int bx = blockIdx.x * TILE_DIM;
  const int by = blockIdx.y * TILE_DIM;
  
  // ============ STEP 1: LOAD TILE WITH HALO ============
  // We need to load TILE_DIM + 2*radius rows
  const int tile_height = TILE_DIM + 2 * radius;
  
  for (int local_row = ty; local_row < tile_height; local_row += TILE_DIM) {
    for (int local_col = tx; local_col < TILE_DIM; local_col += TILE_DIM) {
      int global_row = by + local_row - radius;
      int global_col = bx + local_col;
      
      float val = 0.0f;
      if (global_row >= 0 && global_row < nrows && global_col >= 0 && global_col < ncols) {
        val = imgin[global_row * ncols + global_col];
      }
      s_tile[local_row * TILE_DIM + local_col] = val;
    }
  }
  __syncthreads();
  
  // ============ STEP 2: TRANSPOSE IN SHARED MEMORY ============
  // Transpose: rows become columns
  // After transpose, what was a vertical strip is now a horizontal strip
  if (tx < TILE_DIM && ty < TILE_DIM) {
    for (int k = 0; k < tile_height; k++) {
      int src_idx = k * TILE_DIM + ty;
      int dst_idx = ty * tile_height + k;
      s_conv[dst_idx] = s_tile[src_idx];
    }
  }
  __syncthreads();
  
  // ============ STEP 3: HORIZONTAL CONVOLUTION ON TRANSPOSED DATA ============
  if (tx < TILE_DIM && ty < TILE_DIM) {
    int conv_col = ty;
    int conv_row = tx;
    
    // Check boundaries (remember dimensions are swapped)
    if (conv_col < radius || conv_col >= TILE_DIM - radius) {
      // Will write zero later
    } else {
      float sum = 0.0f;
      int center = conv_row * tile_height + conv_col + radius;
      
      #pragma unroll
      for (int k = 0; k < kernel_width && k < 15; k++) {
        sum += s_conv[center - radius + k] * c_kernel[k];
      }
      
      // Store result back in s_tile (reusing memory)
      s_tile[tx * TILE_DIM + ty] = sum;
    }
  }
  __syncthreads();
  
  // ============ STEP 4: WRITE RESULT ============
  int global_row = by + ty;
  int global_col = bx + tx;
  
  if (global_row < nrows && global_col < ncols) {
    // Apply boundary conditions
    if (global_row < radius || global_row >= nrows - radius) {
      imgout[global_row * ncols + global_col] = 0.0f;
    } else {
      imgout[global_row * ncols + global_col] = s_tile[ty * TILE_DIM + tx];
    }
  }
}
 
 /*********************************************************************
  * Host Wrapper Functions
  *********************************************************************/
static void _convolveImageHoriz(
  _KLT_FloatImage imgin,
  ConvolutionKernel kernel,
  _KLT_FloatImage imgout)
{
  const int ncols = imgin->ncols;
  const int nrows = imgin->nrows;
  const size_t nbytes = ncols * nrows * sizeof(float);
  
  ensure_gpu_buffers(nbytes);
  
  // Copy kernel to constant memory (reversed to match CPU convention)
  // CPU applies kernel in reverse: kernel.data[width-1] at left, kernel.data[0] at right
  // GPU applies forward: c_kernel[0] at left, c_kernel[width-1] at right
  float reversed_kernel[MAX_KERNEL_SIZE];
  for (int i = 0; i < kernel.width; i++) {
    reversed_kernel[i] = kernel.data[kernel.width - 1 - i];
  }
  CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_kernel), reversed_kernel, 
    kernel.width * sizeof(float), 0, hipMemcpyHostToDevice, g_gpu.stream));
   
   // Copy input to device
   CUDA_CHECK(hipMemcpyAsync(g_gpu.d_img1, imgin->data, nbytes,
     hipMemcpyHostToDevice, g_gpu.stream));
   
   // Launch configuration
   const int radius = kernel.width / 2;
   dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
   dim3 grid((ncols + BLOCK_DIM_X - 1) / BLOCK_DIM_X,
             (nrows + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);
   
  // Shared memory size (must match kernel calculation!)
  const int tile_stride = BLOCK_DIM_X + 2 * radius + 8;  // +8 for padding
  size_t shared_bytes = BLOCK_DIM_Y * tile_stride * sizeof(float);
   
   // Enable 64KB shared memory if needed (T4 supports it)
   if (shared_bytes > 48 * 1024) {
     CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(convolveHoriz_Optimized),
       hipFuncAttributeMaxDynamicSharedMemorySize, 64 * 1024));
     CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(convolveHoriz_Optimized),
       hipFuncAttributePreferredSharedMemoryCarveout, 100)); // 64KB shared
   }
   
   convolveHoriz_Optimized<<<grid, block, shared_bytes, g_gpu.stream>>>(
     g_gpu.d_img1, g_gpu.d_img2, ncols, nrows, kernel.width);
   
   CUDA_CHECK(hipGetLastError());
   
   // Copy result back
   CUDA_CHECK(hipMemcpyAsync(imgout->data, g_gpu.d_img2, nbytes,
     hipMemcpyDeviceToHost, g_gpu.stream));
   
   CUDA_CHECK(hipStreamSynchronize(g_gpu.stream));
   
   imgout->ncols = ncols;
   imgout->nrows = nrows;
 }
 
static void _convolveImageVert(
  _KLT_FloatImage imgin,
  ConvolutionKernel kernel,
  _KLT_FloatImage imgout)
{
  const int ncols = imgin->ncols;
  const int nrows = imgin->nrows;
  const size_t nbytes = ncols * nrows * sizeof(float);
  
  ensure_gpu_buffers(nbytes);
  
  // Copy kernel to constant memory (reversed to match CPU convention)
  float reversed_kernel[MAX_KERNEL_SIZE];
  for (int i = 0; i < kernel.width; i++) {
    reversed_kernel[i] = kernel.data[kernel.width - 1 - i];
  }
  CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_kernel), reversed_kernel,
    kernel.width * sizeof(float), 0, hipMemcpyHostToDevice, g_gpu.stream));
  
  // Copy input to device
  CUDA_CHECK(hipMemcpyAsync(g_gpu.d_img1, imgin->data, nbytes,
    hipMemcpyHostToDevice, g_gpu.stream));
  
  // ============ SINGLE FUSED KERNEL: TRANSPOSE + CONVOLVE + TRANSPOSE ============
  const int radius = kernel.width / 2;
  dim3 block(TILE_DIM, TILE_DIM);
  dim3 grid((ncols + TILE_DIM - 1) / TILE_DIM,
            (nrows + TILE_DIM - 1) / TILE_DIM);
  
  // Calculate shared memory: two tiles (one for load, one for transposed convolution)
  const int tile_height = TILE_DIM + 2 * radius;
  size_t shared_bytes = (tile_height * TILE_DIM + tile_height * (TILE_DIM + 2*radius + 8)) * sizeof(float);
  
  if (shared_bytes > 48 * 1024) {
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(convolveVert_FusedTranspose),
      hipFuncAttributeMaxDynamicSharedMemorySize, 64 * 1024));
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(convolveVert_FusedTranspose),
      hipFuncAttributePreferredSharedMemoryCarveout, 100));
  }
  
  // Single kernel does everything!
  convolveVert_FusedTranspose<<<grid, block, shared_bytes, g_gpu.stream>>>(
    g_gpu.d_img1, g_gpu.d_img2, ncols, nrows, kernel.width);
  
  CUDA_CHECK(hipGetLastError());
  
  // Copy result back to host
  CUDA_CHECK(hipMemcpyAsync(imgout->data, g_gpu.d_img2, nbytes,
    hipMemcpyDeviceToHost, g_gpu.stream));
  
  CUDA_CHECK(hipStreamSynchronize(g_gpu.stream));
  
  imgout->ncols = ncols;
  imgout->nrows = nrows;
}
 
/*********************************************************************
 * Separable Convolution - OPTIMIZED GPU VERSION
 * 
 * Keep data on GPU for both passes - only 2 CPU↔GPU transfers total!
 *********************************************************************/
static void _convolveSeparate(
  _KLT_FloatImage imgin,
  ConvolutionKernel horiz_kernel,
  ConvolutionKernel vert_kernel,
  _KLT_FloatImage imgout)
{
  const int ncols = imgin->ncols;
  const int nrows = imgin->nrows;
  const size_t nbytes = ncols * nrows * sizeof(float);
  
  ensure_gpu_buffers(nbytes);
  
  // ============ UPLOAD INPUT ONCE ============
  CUDA_CHECK(hipMemcpyAsync(g_gpu.d_img1, imgin->data, nbytes,
    hipMemcpyHostToDevice, g_gpu.stream));
  
  // ============ HORIZONTAL PASS (GPU → GPU) ============
  {
    // Copy kernel to constant memory (reversed)
    float reversed_kernel[MAX_KERNEL_SIZE];
    for (int i = 0; i < horiz_kernel.width; i++) {
      reversed_kernel[i] = horiz_kernel.data[horiz_kernel.width - 1 - i];
    }
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_kernel), reversed_kernel,
      horiz_kernel.width * sizeof(float), 0, hipMemcpyHostToDevice, g_gpu.stream));
    
    const int radius = horiz_kernel.width / 2;
    dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 grid((ncols + BLOCK_DIM_X - 1) / BLOCK_DIM_X,
              (nrows + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);
    
    const int tile_stride = BLOCK_DIM_X + 2 * radius + 8;
    size_t shared_bytes = BLOCK_DIM_Y * tile_stride * sizeof(float);
    
    if (shared_bytes > 48 * 1024) {
      CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(convolveHoriz_Optimized),
        hipFuncAttributeMaxDynamicSharedMemorySize, 64 * 1024));
      CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(convolveHoriz_Optimized),
        hipFuncAttributePreferredSharedMemoryCarveout, 100));
    }
    
    // d_img1 → d_img2
    convolveHoriz_Optimized<<<grid, block, shared_bytes, g_gpu.stream>>>(
      g_gpu.d_img1, g_gpu.d_img2, ncols, nrows, horiz_kernel.width);
    
    CUDA_CHECK(hipGetLastError());
  }
  
  // ============ VERTICAL PASS (GPU → GPU) ============
  {
    // Copy kernel to constant memory (reversed)
    float reversed_kernel[MAX_KERNEL_SIZE];
    for (int i = 0; i < vert_kernel.width; i++) {
      reversed_kernel[i] = vert_kernel.data[vert_kernel.width - 1 - i];
    }
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_kernel), reversed_kernel,
      vert_kernel.width * sizeof(float), 0, hipMemcpyHostToDevice, g_gpu.stream));
    
    const int radius = vert_kernel.width / 2;
    dim3 block(TILE_DIM, TILE_DIM);
    dim3 grid((ncols + TILE_DIM - 1) / TILE_DIM,
              (nrows + TILE_DIM - 1) / TILE_DIM);
    
    const int tile_height = TILE_DIM + 2 * radius;
    size_t shared_bytes = (tile_height * TILE_DIM + tile_height * (TILE_DIM + 2*radius + 8)) * sizeof(float);
    
    if (shared_bytes > 48 * 1024) {
      CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(convolveVert_FusedTranspose),
        hipFuncAttributeMaxDynamicSharedMemorySize, 64 * 1024));
      CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(convolveVert_FusedTranspose),
        hipFuncAttributePreferredSharedMemoryCarveout, 100));
    }
    
    // d_img2 → d_img1
    convolveVert_FusedTranspose<<<grid, block, shared_bytes, g_gpu.stream>>>(
      g_gpu.d_img2, g_gpu.d_img1, ncols, nrows, vert_kernel.width);
    
    CUDA_CHECK(hipGetLastError());
  }
  
  // ============ DOWNLOAD RESULT ONCE ============
  CUDA_CHECK(hipMemcpyAsync(imgout->data, g_gpu.d_img1, nbytes,
    hipMemcpyDeviceToHost, g_gpu.stream));
  
  CUDA_CHECK(hipStreamSynchronize(g_gpu.stream));
  
  imgout->ncols = ncols;
  imgout->nrows = nrows;
}
 
 /*********************************************************************
  * Kernel Computation (unchanged from original)
  *********************************************************************/
 static void _computeKernels(
   float sigma,
   ConvolutionKernel *gauss,
   ConvolutionKernel *gaussderiv)
 {
   const float factor = 0.01f;
   int i;
 
   assert(MAX_KERNEL_WIDTH % 2 == 1);
   assert(sigma >= 0.0);
 
   {
     const int hw = MAX_KERNEL_WIDTH / 2;
     float max_gauss = 1.0f, max_gaussderiv = (float)(sigma*exp(-0.5f));
   
     for (i = -hw; i <= hw; i++) {
       gauss->data[i+hw] = (float)exp(-i*i / (2*sigma*sigma));
       gaussderiv->data[i+hw] = -i * gauss->data[i+hw];
     }
 
     gauss->width = MAX_KERNEL_WIDTH;
     for (i = -hw; fabs(gauss->data[i+hw] / max_gauss) < factor; 
          i++, gauss->width -= 2);
     gaussderiv->width = MAX_KERNEL_WIDTH;
     for (i = -hw; fabs(gaussderiv->data[i+hw] / max_gaussderiv) < factor; 
          i++, gaussderiv->width -= 2);
     if (gauss->width == MAX_KERNEL_WIDTH || 
         gaussderiv->width == MAX_KERNEL_WIDTH)
       KLTError("(_computeKernels) MAX_KERNEL_WIDTH %d is too small for "
                "a sigma of %f", MAX_KERNEL_WIDTH, sigma);
   }
 
   for (i = 0; i < gauss->width; i++)
     gauss->data[i] = gauss->data[i+(MAX_KERNEL_WIDTH-gauss->width)/2];
   for (i = 0; i < gaussderiv->width; i++)
     gaussderiv->data[i] = gaussderiv->data[i+(MAX_KERNEL_WIDTH-gaussderiv->width)/2];
 
   {
     const int hw = gaussderiv->width / 2;
     float den;
       
     den = 0.0;
     for (i = 0; i < gauss->width; i++) den += gauss->data[i];
     for (i = 0; i < gauss->width; i++) gauss->data[i] /= den;
     den = 0.0;
     for (i = -hw; i <= hw; i++) den -= i*gaussderiv->data[i+hw];
     for (i = -hw; i <= hw; i++) gaussderiv->data[i+hw] /= den;
   }
 
   sigma_last = sigma;
 }
 
 /*********************************************************************
  * Public API Functions
  *********************************************************************/
 void _KLTToFloatImage(
   KLT_PixelType *img,
   int ncols, int nrows,
   _KLT_FloatImage floatimg)
 {
   KLT_PixelType *ptrend = img + ncols*nrows;
   float *ptrout = floatimg->data;
 
   assert(floatimg->ncols >= ncols);
   assert(floatimg->nrows >= nrows);
 
   floatimg->ncols = ncols;
   floatimg->nrows = nrows;
 
   while (img < ptrend) *ptrout++ = (float)*img++;
 }
 
 void _KLTGetKernelWidths(
   float sigma,
   int *gauss_width,
   int *gaussderiv_width)
 {
   _computeKernels(sigma, &gauss_kernel, &gaussderiv_kernel);
   *gauss_width = gauss_kernel.width;
   *gaussderiv_width = gaussderiv_kernel.width;
 }
 
 void _KLTComputeGradients(
   _KLT_FloatImage img,
   float sigma,
   _KLT_FloatImage gradx,
   _KLT_FloatImage grady)
 {
   assert(gradx->ncols >= img->ncols);
   assert(gradx->nrows >= img->nrows);
   assert(grady->ncols >= img->ncols);
   assert(grady->nrows >= img->nrows);
 
   if (fabs(sigma - sigma_last) > 0.05)
     _computeKernels(sigma, &gauss_kernel, &gaussderiv_kernel);
   
   ensure_gpu_buffers(img->ncols * img->nrows * sizeof(float));
   
   _convolveSeparate(img, gaussderiv_kernel, gauss_kernel, gradx);
   _convolveSeparate(img, gauss_kernel, gaussderiv_kernel, grady);
 }
 
 void _KLTComputeSmoothedImage(
   _KLT_FloatImage img,
   float sigma,
   _KLT_FloatImage smooth)
 {
   assert(smooth->ncols >= img->ncols);
   assert(smooth->nrows >= img->nrows);
 
   if (fabs(sigma - sigma_last) > 0.05)
     _computeKernels(sigma, &gauss_kernel, &gaussderiv_kernel);
 
   ensure_gpu_buffers(img->ncols * img->nrows * sizeof(float));
   
   _convolveSeparate(img, gauss_kernel, gauss_kernel, smooth);
 }
 
// Cleanup function (call at program exit)
void _KLTCleanupGPU() {
  if (g_gpu.initialized) {
    if (g_gpu.d_img1) hipFree(g_gpu.d_img1);
    if (g_gpu.d_img2) hipFree(g_gpu.d_img2);
    hipStreamDestroy(g_gpu.stream);
    g_gpu.initialized = false;
  }
}
